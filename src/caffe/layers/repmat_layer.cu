#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/sds_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void RepMatForward(const int nthreads, const Dtype* bottom_data, Dtype* top_data, 
const int top_n, const int top_c, const int top_h, const int top_w, 
const int bottom_n, const int bottom_c, const int bottom_h, const int bottom_w) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = (index % top_w) % bottom_w;
    int h = ((index / top_w) % top_h) % bottom_h;     
    int c = (((index / top_w) / top_h) % top_c) % bottom_c;
    int n = (((index / top_w) / top_h) / top_c) % bottom_n;
    top_data[index] = bottom_data[ w + bottom_w*(h + bottom_h*(c + bottom_c*n)) ];





  }

}

template <typename Dtype>
void RepMatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
 
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  RepMatForward<Dtype><<<CAFFE_GET_BLOCKS(top[0]->count()),CAFFE_CUDA_NUM_THREADS>>>(
      top[0]->count(), bottom_data, top_data, top_n, top_c, top_h, top_w, 
      bottom_n, bottom_c, bottom_h, bottom_w); 
}


template <typename Dtype>
__global__ void RepMatBackward(const int nthreads, const Dtype* top_diff, Dtype* bottom_diff, 
const int top_n, const int top_c, const int top_h, const int top_w, 
const int bottom_n, const int bottom_c, const int bottom_h, const int bottom_w,
int rep_n, int rep_c, int rep_h, int rep_w) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % bottom_w;
    int h = (index / bottom_w) % bottom_h;     
    int c = ((index / bottom_w) / bottom_h) % bottom_c;
    int n = ((index / bottom_w) / bottom_h) / bottom_c;
    for(int n_o = 0; n_o<rep_n; n_o++){
      for(int c_o = 0; c_o<rep_c; c_o++){
        for(int h_o = 0; h_o<rep_h; h_o++){
          for(int w_o = 0; w_o<rep_w; w_o++){
            int offset = (((n_o*bottom_n + n)*top_c + c_o*bottom_c + c)*top_h + h_o*bottom_h + h)*top_w + w_o*bottom_w + w;
            bottom_diff[index] += top_diff[offset];
          }
        }
      }
    }



  }

}


template <typename Dtype>
void RepMatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) { return; }
 
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  caffe_gpu_set(bottom[0]->count(), static_cast<Dtype>(0.0), bottom_diff);
  RepMatBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()),CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->count(), top_diff, bottom_diff, top_n, top_c, top_h, top_w, 
      bottom_n, bottom_c, bottom_h, bottom_w,
      rep_n, rep_c, rep_h, rep_w); 

//Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(RepMatLayer);

}  // namespace caffe
